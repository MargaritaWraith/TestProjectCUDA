#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#include <stdio.h>
#include <ctime>
#include <clocale>


hipError_t mulWithCuda(int *c, const int *a, const int *b, unsigned int size);

// Точка входа в GPU
__global__ void mulKernel(int *c, const int *a, const int *b)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = a[i] * b[i];
	//printf("{1,2,3,4,5} * {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n", //можно вывести на экран средствами GPU
	//	c[0], c[1], c[2], c[3], c[4]);
}

// Точка входа в приложение
int main()
{
	setlocale(LC_CTYPE, "rus");
	
	const int arraySize = 8000;
	int a[arraySize] = { 0 };
	int b[arraySize] = { 0 };
	int c[arraySize] = { 0 };

	for (int i = 0; i < arraySize; i++)
	{
		a[i] = i + 1;
		b[i] = a[i] * 10;
	}

	srand(time(0));

	// Add vectors in parallel.
	hipError_t cudaStatus = mulWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "mulWithCuda failed!");
		return 1;
	}

	printf("Время работы метода: %d мс \n", clock());


	printf("\n*******************************************\n");
	printf("Вывод первых и последних 10 результатов перемножения массивов размерностью 8000 элементов: \n\n");
	printf("{1,2,3,4,5,6,7,8,9,10} * {10,20,30,40,50,60,70,80,90,100} = {%d,%d,%d,%d,%d,%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4], c[5], c[6], c[7], c[8], c[9]);
	printf("{7991,7992,7993,7994,7995,7996,7997,7998,7999,8000} * {79910,79920,79930,79940,79950,79960,79970,79980,79990,80000} = {%d,%d,%d,%d,%d,%d,%d,%d,%d,%d}\n",
		c[7990], c[7991], c[7992], c[7993], c[7994], c[7995], c[7996], c[7997], c[7998], c[7999]);

	printf("\n*******************************************\n\n");

	printf("Основные данные по устройству: \n\n");

	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	for (int device = 0; device < deviceCount; device++) {
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, device);
		printf("Номер устройства: %d\n", device);
		printf("Имя устройства: %s\n", deviceProp.name);
		printf("Объем глобальной памяти: %d\n", deviceProp.totalGlobalMem);
		printf("Объем shared-памяти в блоке : %d\n", deviceProp.sharedMemPerBlock);
		printf("Объем регистровой памяти: %d\n", deviceProp.regsPerBlock);
		printf("Размер warp'a: %d\n", deviceProp.warpSize);
		printf("Размер шага памяти: %d\n", deviceProp.memPitch);
		printf("Макс количество потоков в блоке: %d\n", deviceProp.maxThreadsPerBlock);

		printf("Максимальная размерность потока: x = %d, y = %d, z = %d\n",
			deviceProp.maxThreadsDim[0],
			deviceProp.maxThreadsDim[1],
			deviceProp.maxThreadsDim[2]);

		printf("Максимальный размер сетки: x = %d, y = %d, z = %d\n",
			deviceProp.maxGridSize[0],
			deviceProp.maxGridSize[1],
			deviceProp.maxGridSize[2]);

		printf("Тактовая частота: %d\n", deviceProp.clockRate);
		printf("Общий объем константной памяти: %d\n", deviceProp.totalConstMem);
		printf("Вычислительная мощность: %d.%d\n", deviceProp.major, deviceProp.minor);
		printf("Величина текстурного выравнивания : %d\n", deviceProp.textureAlignment);
		printf("Количество процессоров: %d\n", deviceProp.multiProcessorCount);
	}
	printf("\n*******************************************\n\n");

	printf("Расчёт на хосте\n");
	srand(time(0));

	for (int i = 0; i < arraySize; i++)
	{
		c[i] = a[i] * b[i];
	}
	printf("Время работы метода на хосте: %d мс \n", clock());
	printf("Вывод первых и последних 10 результатов перемножения массивов размерностью 8000 элементов: \n\n");
	printf("{1,2,3,4,5,6,7,8,9,10} * {10,20,30,40,50,60,70,80,90,100} = {%d,%d,%d,%d,%d,%d,%d,%d,%d,%d}\n",
		c[0], c[1], c[2], c[3], c[4], c[5], c[6], c[7], c[8], c[9]);
	printf("{7991,7992,7993,7994,7995,7996,7997,7998,7999,8000} * {79910,79920,79930,79940,79950,79960,79970,79980,79990,80000} = {%d,%d,%d,%d,%d,%d,%d,%d,%d,%d}\n",
		c[7990], c[7991], c[7992], c[7993], c[7994], c[7995], c[7996], c[7997], c[7998], c[7999]);

	printf("\n*******************************************\n\n");

			   
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t mulWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0; // dev - находится на GPU
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0); // указываем, что работаем на "0"-й карте, м.б. несколько
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int)); // выделяем память на переменную
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int)); // выделяем память на переменную
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int)); // выделяем память на переменную
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice); // копируем значения переменной с хоста на GPU 
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice); // копируем значения переменной с хоста на GPU 
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	
	// Launch a kernel on the GPU with one thread for each element.
	dim3 block(64, 1);
	dim3 grid((size / 64), 1);
	mulKernel << <grid, block >> > (dev_c, dev_a, dev_b); // запуск функции с параметрами (size - размер массива)
	   

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "mulKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching mulKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
